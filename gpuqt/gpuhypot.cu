#include "hip/hip_runtime.h"


#include <stdio.h>
#include <sys/time.h>

void cpuhypot(int *CPU_idata, int *CPU_odata, size_t Totsize);
hipError_t gpuhypot(int *CPU_idata, int *CPU_odata, size_t Totsize, float* Runtimes);

#define BLOCK_SIZE 128
#define NUM_BLOCKS 32768
const int arraySize = NUM_BLOCKS*BLOCK_SIZE;  // Must be a power 2 times BLOCK_SIZE (this code cannot handle other cases)

int *CPU_InputArray;
int *CPU_OutputArray;

__global__ void hypotKernelG(int *GPU_i, int *GPU_o)
{
  // unsigned int  tid = threadIdx.x;                        // gets index of thread in block
  unsigned int  i = blockIdx.x*blockDim.x + threadIdx.x;  // Gets the global position of this thread

  float a,b,x;
  int II;
  
  II = i << 1;
  a = (float)GPU_i[II];
  b = (float)GPU_i[II+1];
  x = sqrt( a*a + b*b );
  GPU_o[i] = (int)x;  // Note: CPU/GPU results may be different in the case of overflow.
}

int main()
{
  float GPURuntimes[4];         // Run times of the GPU code
  struct timeval st, et;
  double StartTime, EndTime;
  hipError_t cudaStatus;
  
  // Create CPU memory to store the input and output arrays
  CPU_InputArray  = (int*)malloc(arraySize*sizeof(int));
  if(CPU_InputArray == NULL) {
    fprintf(stderr,"OOPS. Can't create InputArray using malloc() ...\n\n");
    return EXIT_FAILURE;
  }
  CPU_OutputArray = (int*)malloc(arraySize*sizeof(int)/2);
  if(CPU_OutputArray == NULL) {
    free(CPU_InputArray);
    fprintf(stderr,"OOPS. Can't create OutputArray using malloc() ...\n\n");
    return EXIT_FAILURE;
  }
  
  for (int i = 0; i < arraySize; i++)	{
    CPU_InputArray[i] = (int) rand()*10000000;  // create random numbers from 0 to some big value
  }
  
  // Run it in the CPU to get the gold copy
  gettimeofday(&st, NULL);
  cpuhypot(CPU_InputArray, CPU_OutputArray, arraySize/2);
  gettimeofday(&et, NULL);
  StartTime = st.tv_sec*1000.00 + (st.tv_usec/1000.0);
  EndTime   = et.tv_sec*1000.00 + (et.tv_usec/1000.0);
  printf("Elapsed time = %ld ms\n", (long) (EndTime - StartTime));

  // Compute hypot in parallel.
  cudaStatus = gpuhypot(CPU_InputArray, CPU_OutputArray, arraySize, GPURuntimes);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "\n gpuhypot failed!\n");
    free(CPU_InputArray);
    free(CPU_OutputArray);
    hipDeviceReset();
    return EXIT_FAILURE;
  }

  printf("\nKERNEL = hypotKernelG ...\n");
  printf("Tfr CPU->GPU = %5.2f ms ... \nExecution = %5.2f ms ... \nTfr GPU->CPU = %5.2f ms   \n Total=%5.2f ms\n",
	 GPURuntimes[1], GPURuntimes[2], GPURuntimes[3], GPURuntimes[0]);
  printf("-----------------------------------------------------------------\n");

  // cudaDeviceReset must be called before exiting in order for profiling and
  // tracing tools such as Parallel Nsight and Visual Profiler to show complete traces.
  cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceReset failed!");
    free(CPU_InputArray);
    free(CPU_OutputArray);
    return EXIT_FAILURE;
  }
  
  free(CPU_InputArray);
  free(CPU_OutputArray);
  return EXIT_SUCCESS;
}

void cpuhypot(int *CPU_idata, int *CPU_odata, size_t Totsize)
{
  int n;
  float a,b,x;
  int II;
  
  for(n=0; n<Totsize; n++){
    II = n << 1;
    a = (float)CPU_idata[II];
    b = (float)CPU_idata[II+1];
    x = sqrtf(a*a + b*b);
    CPU_odata[n] = (int) x;
  }
}

// Helper function for using CUDA to compute hypot in parallel.
hipError_t gpuhypot(int *CPU_idata, int *CPU_odata, size_t Totsize, float* Runtimes)
{
  hipEvent_t time1, time2, time3, time4;
  int TotalGPUSize;
  
  int *GPU_idata = 0;
  int *GPU_odata = 0;
  //int blocksize;
  //int totalBlocks = Totsize/BLOCK_SIZE;
  
  // Choose which GPU to run on, change this on a multi-GPU system.
  hipError_t cudaStatus;
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    goto Error;
  }
  
  hipEventCreate(&time1);
  hipEventCreate(&time2);
  hipEventCreate(&time3);
  hipEventCreate(&time4);
  
  hipEventRecord(time1, 0);
  // Allocate GPU buffer for inputs and outputs (hypotenuse)
  TotalGPUSize=Totsize *sizeof(int);
  
  cudaStatus = hipMalloc((void**)&GPU_idata, TotalGPUSize);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }
  
  cudaStatus = hipMalloc((void**)&GPU_odata, TotalGPUSize/2);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }
  
  // Copy input vectors from host memory to GPU buffers.
  cudaStatus = hipMemcpy(GPU_idata, CPU_idata, TotalGPUSize, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }
  
  hipEventRecord(time2, 0);
  // Launch a kernel on the GPU with one thread for each element.
  
  hypotKernelG<<<NUM_BLOCKS/2, BLOCK_SIZE>>>(GPU_idata, GPU_odata);

  // Check for errors immediately after kernel launch.
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "error code %d (%s) launching kernel!\n",
	      cudaStatus, hipGetErrorString(cudaStatus));
      goto Error;
    }

  // cudaDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching kernel!\n",
	    cudaStatus, hipGetErrorString(cudaStatus));
    goto Error;
  }
  
  hipEventRecord(time3, 0);
  // Copy output (results) from GPU buffer to host (CPU) memory.
  cudaStatus = hipMemcpy(CPU_odata, GPU_odata, TotalGPUSize/2, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }
  
  hipEventRecord(time4, 0);
  hipEventSynchronize(time1);
  hipEventSynchronize(time2);
  hipEventSynchronize(time3);
  hipEventSynchronize(time4);
  
  float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime;
  
  hipEventElapsedTime(&totalTime, time1, time4);
  hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
  hipEventElapsedTime(&kernelExecutionTime, time2, time3);
  hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);
  
  Runtimes[0] = totalTime;
  Runtimes[1] = tfrCPUtoGPU;
  Runtimes[2] = kernelExecutionTime;
  Runtimes[3] = tfrGPUtoCPU;

 Error:
  hipFree(GPU_odata);
  hipFree(GPU_idata);
  hipEventDestroy(time1);
  hipEventDestroy(time2);
  hipEventDestroy(time3);
  hipEventDestroy(time4);
  
  return cudaStatus;
}
