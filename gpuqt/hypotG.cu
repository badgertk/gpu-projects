#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <time.h>
#include <Windows.h>


hipError_t gpuhypot(int *CPU_idata, int *CPU_odata, size_t Totsize, float* Runtimes);

#define BLOCK_SIZE	 256

const int arraySize = 32768*BLOCK_SIZE; //Must be a power 2 times BLOCK_SIZE (this code cannot handle other cases)

int *CPU_InputArray;
int *CPU_OutputArray;



__global__ void hypotKernelG(int *GPU_i, int *GPU_o)
{
	unsigned int  tid = threadIdx.x;			// gets index of thread in block
	unsigned int  bid = blockIdx.x*blockDim.x;  // gets index of the block
	unsigned int  i = bid+tid;					// global index of this thread
	int           a,b;                          // temp variables for this thread to use

	a = GPU_i[2*i];
	b = GPU_i[2*i+1];
    GPU_o[i] = sqrt( (double) (a*a + b*b) );
}



int main()
{
	float GPURuntimes[4];         // Run times of the GPU code
	clock_t CPUStartTime, CPUEndTime, CPUElapsedTime;
	hipError_t cudaStatus;
	int InputArraySize,OutputArraySize;
	char key;

	// Create CPU memory to store the input and output arrays
	InputArraySize=arraySize*sizeof(int);
	OutputArraySize=arraySize*sizeof(int)/2;
	CPU_InputArray  = (int*)malloc(InputArraySize);
	if(CPU_InputArray == NULL){ fprintf(stderr,"OOPS. Can't create InputArray using malloc() ...\n\n"); exit(EXIT_FAILURE); }
	CPU_OutputArray = (int*)malloc(OutputArraySize);
	if(CPU_OutputArray == NULL){ fprintf(stderr,"OOPS. Can't create OutputArray using malloc() ...\n\n"); exit(EXIT_FAILURE); }

	for (int i = 0; i < arraySize; i++)	{
		CPU_InputArray[i] = (int) rand()*10000000;       // create random numbers from 0 to some big value
	}

	cudaStatus = gpuhypot(CPU_InputArray, CPU_OutputArray, arraySize, GPURuntimes);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n gpuhypot failed!");
		key=getc(stdin);
		free(CPU_InputArray);
		free(CPU_OutputArray);
		return 1;
	}
	printf("\nKERNEL = hypotKernelG ...\n");
	printf("Tfr CPU->GPU = %5.2f ms ... \nExecution = %5.2f ms ... \nTfr GPU->CPU = %5.2f ms   \n Total=%5.2f ms\n\n",GPURuntimes[1], GPURuntimes[2], GPURuntimes[3], GPURuntimes[0]);
	printf("Tfr CPU->GPU = %5.2f ms  ...  %6d MB  ...  %6.3f GB/s\n",GPURuntimes[1],InputArraySize/1024/1024,(float)InputArraySize/(GPURuntimes[1]*1024.0*1024.0));
	printf("Tfr GPU->CPU = %5.2f ms  ...  %6d MB  ...  %6.3f GB/s\n",GPURuntimes[3],OutputArraySize/1024/1024,(float)OutputArraySize/(GPURuntimes[3]*1024.0*1024.0));
	printf("--------------------------------------------------------------------\n");
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Parallel Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
		key=getc(stdin);
	    free(CPU_InputArray);
	    free(CPU_OutputArray);
        return 1;
    }

	key=getc(stdin);   // wait for a char, so the terminal window doesn't close

    free(CPU_InputArray);
	free(CPU_OutputArray);
    return 0;
}



// Helper function for using CUDA to add vectors in parallel.
hipError_t gpuhypot(int *CPU_idata, int *CPU_odata, size_t Totsize, float* Runtimes)
{
	hipEvent_t time1, time2, time3, time4;
	int TotalGPUSize;

    // Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	hipEventCreate(&time1);
	hipEventCreate(&time2);
	hipEventCreate(&time3);
	hipEventCreate(&time4);

    int *GPU_idata = 0;
    int *GPU_odata = 0;
    

	hipEventRecord(time1, 0);
    // Allocate GPU buffer for inputs and outputs (hypotenuse)
	TotalGPUSize=Totsize *sizeof(int);

    cudaStatus = hipMalloc((void**)&GPU_idata, TotalGPUSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&GPU_odata, TotalGPUSize/2);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(GPU_idata, CPU_idata, TotalGPUSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	hipEventRecord(time2, 0);
    // Launch a kernel on the GPU with one thread for each element.
	int blocksize = BLOCK_SIZE;
	int tempSize = Totsize;
	int totalBlocks = tempSize/BLOCK_SIZE;
	int *tempOut;
	
	hypotKernelG<<<16384,256>>>(GPU_idata, GPU_odata);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}


	hipEventRecord(time3, 0);
    // Copy output (results) from GPU buffer to host (CPU) memory.
    cudaStatus = hipMemcpy(CPU_odata, GPU_odata, TotalGPUSize/2, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	hipEventRecord(time4, 0);
	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime;

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);


	Runtimes[0] = totalTime;
	Runtimes[1] = tfrCPUtoGPU;
	Runtimes[2] = kernelExecutionTime;
	Runtimes[3] = tfrGPUtoCPU;
Error:
    hipFree(GPU_odata);
    hipFree(GPU_idata);
	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);
    
    return cudaStatus;
}