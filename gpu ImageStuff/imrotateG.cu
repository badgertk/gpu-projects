#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <ctype.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

//CUDA STUFF:
#include "hip/hip_runtime.h"
#include ""

//OpenCV stuff
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
using namespace cv;

hipError_t launch_helper(Mat image, int *CPU_OutputArray, float* Runtimes);

int M; //number of rows in image
int N; //number of columns in image
int NumRot;
int a = 0;
Mat zero;

//ip.Vpixels <--> M
//ip.Hpixels <--> N

// These come from CLI arguments:
int BOX_SIZE;			// ThreadsPerBlock == BOX_SIZE * BOX_SIZE

__global__ void rotate_kernel(uchar *GPU_i, uchar *GPU_o, int M, int N, int i, int j){

	int row = blockIdx.x * blockDim.x + threadIdx.x; //row of image
	int col = blockIdx.y * blockDim.y + threadIdx.y; //column of image
	int idx = row*N + col; //which pixel in full 1D array
	uchar output = GPU_i[idx];


    int h,v,c;
	int row2; //new row of image
	int col2; //new column of image

	double X, Y, newY, newX, ScaleFactor;
	double Diagonal, H, V;
	double RotDegrees = 360 / j * i; //in degrees
	double RotAngle = 2*3.141592/360.000*(double) RotDegrees; //in radians
	//printf("We are rotating %d times and iteration# = %d RotAngle = %g\n", j, i, RotAngle);
	// transpose image coordinates to Cartesian coordinates
	// integer div
	c = col;
	h=N/2; 	//halfway of column pixels
	v=M/2;	//halfway of horizontal pixels
	X=(double)c-(double)h;
	Y=(double)v-(double)row;
	
	// pixel rotation matrix	
	newX = cos(RotAngle) * X - sin(RotAngle) * Y;
	newY= sin (RotAngle) * X + cos(RotAngle) * Y;

	
	// Scale to fit everything in the image box CONFIRMED TO BE CORRECT
	H=(double)N;
	V=(double)M;
	Diagonal=sqrt(H*H+V*V);
	ScaleFactor=(N>M) ? V/Diagonal : H/Diagonal;
	newX=newX*ScaleFactor;
	newY = newY*ScaleFactor;
	
	// convert back from Cartesian to image coordinates
	col2= (int)newX+h;
	row2=v-(int)newY;

	// maps old pixel to new pixel
	int idx2 = row2*N + col2;
	GPU_o[idx2] = output;

}

int main(int argc, char *argv[]){

	float GPURuntimes[4]; 	// run times of the GPU code
	float ExecTotalTime, GPUTotalTime;
	hipError_t cudaStatus;
	char filename[100]; //output file name
	int i;

	int *CPU_OutputArray = (int*) 0; 	// where the GPU should copy the output back to
	
	if (argc != 4){
		printf("Improper usage!\n");
		printf("Usage: %s <input image> <output image> <N rotations>\n", argv[0]);
		exit(EXIT_FAILURE);
	}
	BOX_SIZE = 16;
	NumRot = atoi(argv[3]);
	if (NumRot > 30){
		printf("Number of rotations requested is too high!  Adjusted to 30.\n");
		NumRot = 30;
	}
	for (i = 0; i<NumRot; i++){	
	// Load image:
	Mat image;
	image = imread(argv[1], CV_LOAD_IMAGE_GRAYSCALE);
	if (! image.data){
		fprintf(stderr, "Could not open or find the image.\n");
		exit(EXIT_FAILURE);
	}
	printf("Loaded image '%s', size = %dx%d (dims = %d).\n", argv[1], image.cols, image.rows, image.dims);

	//set up global variables for image size
	M = image.rows;
	N = image.cols;
	//start here????
	// Create CPU memory to store the output;
	/*Mat */zero = Mat(M,N,CV_8UC1, Scalar(255));
	sprintf(filename,"%sAROT%d.png", argv[2], i);
	imwrite(filename,zero);
	
	CPU_OutputArray = (int*) malloc(M*N*sizeof(int));
	if (CPU_OutputArray == NULL){
		fprintf(stderr, "OOPS.  Can't create CPU_OutputArray using malloc() ...\n");
		exit(EXIT_FAILURE);
	}
	
	//run it
	cudaStatus = launch_helper(image, CPU_OutputArray, GPURuntimes);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "launch_helper failed!\n");
		free(CPU_OutputArray);
		exit(EXIT_FAILURE);
	}
	
	printf("-----------------------------------------------------------------\n");
	printf("Tfr CPU->GPU = %5.2f ms ... \nExecution = %5.2f ms ... \nTfr GPU->CPU = %5.2f ms   \nSum of Iteration = %5.2f ms\n",
			GPURuntimes[1], GPURuntimes[2], GPURuntimes[3], GPURuntimes[0]);
	ExecTotalTime += GPURuntimes[0];
	GPUTotalTime += GPURuntimes[2];
	printf("\nGPU Execution Time = %5.2f ms \n", GPUTotalTime);
	printf("Total Execution Time = %5.2f ms\n", ExecTotalTime);
	printf("-----------------------------------------------------------------\n");
	
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipDeviceReset failed!\n");
		free(CPU_OutputArray);
		exit(EXIT_FAILURE);
	}
	
	//save image to disk
	Mat result = Mat(M,N,CV_8UC1, CPU_OutputArray);
	imwrite(filename,result);

	if (!imwrite(filename, result)){
		fprintf(stderr, "couldn't write output to disk!\n");
		free(CPU_OutputArray);
		exit(EXIT_FAILURE);
	}
	
	printf("Saved image '%s', size = %dx%d (dims = %d).\n",
	       //filename.c_str(), result.cols, result.rows, result.dims
	       filename, result.cols, result.rows, result.dims);

	free(CPU_OutputArray);
	}
	exit(EXIT_SUCCESS);
}

hipError_t launch_helper(Mat image, int *CPU_OutputArray, float* Runtimes){
	
	hipEvent_t time1, time2, time3, time4;
	int TotalGPUSize; // total size of 1 image in bytes
	uchar *GPU_idata;
	uchar *GPU_odata;
	uchar *GPU_zerodata;
	
	dim3 threadsPerBlock;
	dim3 numBlocks;
	
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0); // use the first GPU
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}
	
	hipEventCreate(&time1);
	hipEventCreate(&time2);
	hipEventCreate(&time3);
	hipEventCreate(&time4);

	hipEventRecord(time1, 0);
	
	// Allocate GPU buffer for inputs and outputs:
	TotalGPUSize = M * N * sizeof(uchar);
	cudaStatus = hipMalloc((void**)&GPU_idata, TotalGPUSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&GPU_odata, TotalGPUSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&GPU_zerodata, TotalGPUSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(GPU_odata, zero.data, TotalGPUSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyzero failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(GPU_idata, image.data, TotalGPUSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	hipEventRecord(time2, 0);

	// Launch a kernel on the GPU with one thread for each pixel.
	threadsPerBlock = dim3(BOX_SIZE, BOX_SIZE);
	numBlocks = dim3(M / threadsPerBlock.x, N / threadsPerBlock.y);
	rotate_kernel<<<numBlocks, threadsPerBlock>>>(GPU_idata, GPU_odata, M, N, a, NumRot);
	
	// Check for errors immediately after kernel launch.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "error code %d (%s) launching kernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching addKernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}

	hipEventRecord(time3, 0);

	// Copy output (results) from GPU buffer to host (CPU) memory.
	cudaStatus = hipMemcpy(CPU_OutputArray, GPU_odata, TotalGPUSize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}
	
	hipEventRecord(time4, 0);
	
	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime;

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	Runtimes[0] = totalTime;
	Runtimes[1] = tfrCPUtoGPU;
	Runtimes[2] = kernelExecutionTime;
	Runtimes[3] = tfrGPUtoCPU;

	Error:
	hipFree(GPU_odata);
	hipFree(GPU_idata);
	hipFree(GPU_zerodata);
	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);

	a++;
	return cudaStatus;
}
