#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <ctype.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

//CUDA STUFF:
#include "hip/hip_runtime.h"
#include ""

//OpenCV stuff
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
using namespace cv;

#define BOX_SIZE		16 		//ThreadsPerBlock == BOX_SIZE * BOX_SIZE
#define PI				3.1415926
#define EDGE			0
#define NOEDGE			255

int M; //number of rows in the image
int N; //number of columns in the image
int TotalSize;
int NumIter = 16;
int ThreshLo = 0;
Mat zero;
int stream = 4; //???
int levels = 8; //???

uchar *CPU_InputArray;
uchar *CPU_OutputArray;
int TotalGPUSize;

hipError_t launch_helper(uchar *CPU_InputArray, uchar *CPU_OutputArray, float *GPURuntimes);

hipError_t checkCuda(hipError_t result,int line)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s at line : %d\n", hipGetErrorString(result),line);
    // We should be free()ing CPU+GPU memory here, but we're relying on the OS
    // to do it for us.
    hipDeviceReset();
    assert(result == hipSuccess);
  }
  return result;
}
inline
hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    // We should be free()ing CPU+GPU memory here, but we're relying on the OS
    // to do it for us.
    hipDeviceReset();
    assert(result == hipSuccess);
  }
  return result;
}


//ip.Vpixels <--> M
//ip.Hpixels <--> N

//kernels
__global__ void GaussianFilter(uchar *GPU_i, double *Gauss_o, int M, int N, int offsetx, int offsety){

	int row = blockIdx.x * blockDim.x + threadIdx.x + offsetx; //row of image
	int col = blockIdx.y * blockDim.y + threadIdx.y + offsety; //column of image
	int idx = row*N + col; //which pixel in full 1D array
	//uchar output = GPU_i[idx];
	int i,j;
	double G;
	double Gauss[5][5] = {	{ 2, 4,  5,  4,  2 },
						{ 4, 9,  12, 9,  4 },
						{ 5, 12, 15, 12, 5 },
						{ 4, 9,  12, 9,  4 },
						{ 2, 4,  5,  4,  2 }	};
	
	if ((row < 2) || (row > (M - 3))) goto End;
	//col = 2;
	G = 0.0;
	int idx2, row2, col2;
	for (i=-2; i<=2; i++){
		for (j=-2; j<=2; j++){
			row2 = row + i;
			col2 = col + j;
			idx2 = row2*N + col2;
			G = G + GPU_i[idx2] * Gauss[i + 2][j + 2];	
		}
	}
	
	Gauss_o[idx] = G/ (double)159.00;	
	//printf("Gauss_o[] = %f", Gauss_o[idx]); //looks like the numbers are right
	End:;
}

__global__ void Sobel(double *Gauss_i, double *Gradient_o, double *Theta_o, int M, int N, int offsetx, int offsety){
	int row = blockIdx.x * blockDim.x + threadIdx.x + offsetx; //row of image
	int col = blockIdx.y * blockDim.y + threadIdx.y + offsety; //column of image
	int idx = row*N + col; //which pixel in full 1D array
	//uchar output = Gauss_i[idx]; okay so these numbers are right
	//printf("Gauss_i[] = %f", Gauss_i[idx]);
	int i,j;
	double GX,GY;
	//printf("row = %d, col = %d", row, col);
	
double Gx[3][3] = {		{ -1, 0, 1 },
						{ -2, 0, 2 },
						{ -1, 0, 1 }	};

double Gy[3][3] = {		{ -1, -2, -1 },
						{  0,  0,  0 },
						{  1,  2,  1 }	};
	
	if ((row<1) || (row>(M-2))) goto End;
	//col = 1;
	if (col<=(N-2)){
		GX = 0.0; GY = 0.0;
		int row2, col2, idx2;
		for (i = -1; i <= 1; i++){
			for (j = -1; j<= 1; j++){
				row2 = row + i;
				col2 = col + j;
				//printf("row2 = %d, N = %d, col2 = %d", row2, N, col2);
				idx2 = row2*N + col2; //this is wrong
				GX = GX + Gauss_i[idx2] * Gx[i+1][j+1];
				//printf("Gauss_i[] = %f", Gauss_i[idx2]); //a lot of 124.92
				GY = GY + Gauss_i[idx2] * Gy[i+1][j+1];
				//printf("Gy[] = %f", Gy[i+1][j+1]);
			}
		}
		
		Gradient_o[idx] = sqrt(GX*GX+GY*GY);
		//printf("GX = %f GY = %f Gradient = %f", GX, GY, sqrt(GX*GX+GY*GY)); //GX always = 0 and GY always = 499?
		Theta_o[idx] = atan(GX/GY) * 180.0/PI;
	}
	End:;
}

__global__ void Threshold(double *Gradient_i, double *Theta_i, uchar *GPU_o, int M, int N, int offsetx, int offsety, int ThreshLo, int ThreshHi){ //Threshold values need to be part of the parameters
	int row = blockIdx.x * blockDim.x + threadIdx.x + offsetx; //row of image
	int col = blockIdx.y * blockDim.y + threadIdx.y + offsety; //column of image
	int idx = row*N + col; //which pixel in full 1D array
	//uchar output = Gradient_i[idx];
	
	uchar PIXVAL;
	double L,H,G,T;
	
	if ((row<1) || (row>(M-2))) goto End;
	//col = 1;
	L = (double) ThreshLo; H = (double)ThreshHi;
	G = Gradient_i[idx];
	PIXVAL = NOEDGE;
	if (G <= L){
		PIXVAL = NOEDGE;
	} else if (G >= H){
		//printf("G = %f and H = %f", G, H);
		PIXVAL = EDGE;
	} else{
			//printf("GOT IN HERE?");
		T = Theta_i [idx];
		if ((T < -67.5) || (T > 67.5)){
			//look left and right
			PIXVAL = ((Gradient_i[row*N + col - 1] > H) || (Gradient_i[row*N + col + 1] > H)) ? EDGE:NOEDGE;
		} else if ((T >= -22.5) && (T <= 22.5)){
			//look top and bottom
			PIXVAL = ((Gradient_i[(row - 1)*N + col] > H) || (Gradient_i[(row + 1)*N + col] > H)) ? EDGE:NOEDGE;
		} else if ((T > 22.5) && (T <= 67.5)){
			//look upper right and lower left
			PIXVAL = ((Gradient_i[(row - 1)*N + col + 1] > H) || (Gradient_i[(row + 1)*N + col - 1] > H)) ? EDGE:NOEDGE;
		} else if ((T >= -67.5) && (T < -22.5)){
			//look upper left and lower right
			PIXVAL = ((Gradient_i[(row - 1)*N + col - 1] > H) || (Gradient_i[(row + 1)*N + col + 1] > H)) ? EDGE:NOEDGE;
		}
	}
	//printf("pixval = %d", PIXVAL);
	GPU_o[idx] = 255 - PIXVAL;
	End:;
}

int main(int argc, char *argv[]){
	float GPURuntimes[4]; //run times of the GPU code

	char filename[100]; //output file name
	int j;

	if (argc != 3){
		printf("Improper Usage!\n");
		printf("Usage: %s <input image> <output image>\n", argv[0]);
		exit(EXIT_FAILURE);
	}
	//Load image:
	Mat image;
	image = imread(argv[1], CV_LOAD_IMAGE_GRAYSCALE);
	if (! image.data){
		fprintf(stderr, "Could not open or find the image.\n");
		exit(EXIT_FAILURE);
	}
	printf("Loaded image '%s', size = %dx%d (dims = %d).\n", argv[1], image.cols, image.rows, image.dims);
	
	//set up global variables for image size
	M = image.rows;
	N = image.cols;
	TotalGPUSize = M * N * sizeof(uchar);



	checkCuda(hipHostMalloc( (void**)&CPU_InputArray,  TotalGPUSize), __LINE__); //checkcuda
	memcpy(CPU_InputArray, image.data, TotalGPUSize);  // always the same image
	//  Allocate the output while we're at it:
	checkCuda(hipHostMalloc( (void**)&CPU_OutputArray, TotalGPUSize), __LINE__); //checkcuda

	for (j = 0; j < NumIter; j++){
		//Create CPU memory to store the output;
		zero = Mat(M,N,CV_8UC1, Scalar(255)); //start by making every pixel white
		sprintf(filename, "%s%d.png",argv[2],j);
		imwrite(filename, zero);
	
		checkCuda(launch_helper(CPU_InputArray, CPU_OutputArray, GPURuntimes), __LINE__); //checkcuda

		// FIX THIS LAST
		printf("-----------------------------------------------------------------\n");
		printf("Tfr CPU->GPU = %5.2f ms ... \nExecution = %5.2f ms ... \nTfr GPU->CPU = %5.2f ms   \nSum of Iteration = %5.2f ms\n",
				GPURuntimes[1], GPURuntimes[2], GPURuntimes[3], GPURuntimes[0]);
		//ExecTotalTime += GPURuntimes[0];
		//GPUTotalTime += GPURuntimes[2];
		//printf("\nGPU Execution Time = %5.2f ms \n", GPUTotalTime);
		//printf("Total Execution Time = %5.2f ms\n", ExecTotalTime);
		printf("-----------------------------------------------------------------\n");


	//save image to disk
	Mat result = Mat(M,N,CV_8UC1, CPU_OutputArray);
	imwrite(filename,result);

		

		if (!imwrite(filename, result)){
			fprintf(stderr, "couldn't write output to disk!\n");
	hipHostFree(CPU_InputArray);
    hipHostFree(CPU_OutputArray);	

			exit(EXIT_FAILURE);
		} 
				
		printf("Saved image '%s', size = %dx%d (dims = %d).\n",
			   //filename.c_str(), result.cols, result.rows, result.dims);
			   filename, result.cols, result.rows, result.dims);
  

  }
	hipHostFree(CPU_InputArray);
    hipHostFree(CPU_OutputArray);	 
	hipDeviceReset();

  // Done.
  exit(EXIT_SUCCESS);

} //end of main

// Helper function for launching the CUDA kernel (including memcpy, etc.):
hipError_t launch_helper(uchar *CPU_InputArray, uchar *CPU_OutputArray, float *Runtimes){

		hipEvent_t time1, time2, time3, time4;
		//int ucharGPUSize, doubleGPUSize; // total size of 1 image in bytes
		uchar *GPU_idata;
		uchar *GPU_odata;
		//uchar *GPU_zerodata;
		double *GPU_Gaussdata;
		double *GPU_Gradientdata;
		double *GPU_Thetadata;
		
		dim3 threadsPerBlock;
		dim3 numBlocks;

  	  dim3 sharedBlocks;
  	  int shared_mem_size;
  	  dim3 streamSize;
  	  int UCharTotalSize = (M/levels+4)*N*sizeof(uchar);
	  int DoubleTotalGPUSize = M * N * sizeof(double);

  	threadsPerBlock = dim3(BOX_SIZE,BOX_SIZE);
	numBlocks = dim3(ceil((float)M / threadsPerBlock.x),ceil((float)N / threadsPerBlock.y));
	sharedBlocks = dim3(ceil((float)numBlocks.x/levels),ceil((float)numBlocks.y/stream));
	shared_mem_size = threadsPerBlock.x*threadsPerBlock.y;
	printf("NumThreads/Block: %d, NumBlocks: %d, %d, Shared Blocks: %d, %d\n",threadsPerBlock.x*threadsPerBlock.y,numBlocks.x,numBlocks.y,sharedBlocks.x,sharedBlocks.y);


	hipStream_t streams[stream+1];
	for (int i = 0; i < stream+1; ++i) { //one extra for dummy purposes
		checkCuda(hipStreamCreate(&streams[i]), __LINE__); //checkcuda
	}
	
	hipEventCreate(&time1);
	hipEventCreate(&time2);
	hipEventCreate(&time3);
	hipEventCreate(&time4);
		

    // Allocate GPU buffer for input and output: all of them checkcuda
    hipMalloc((void**)&GPU_idata, TotalGPUSize);
    hipMalloc((void**)&GPU_odata, TotalGPUSize);
    hipMalloc((void**)&GPU_Gaussdata, DoubleTotalGPUSize);
    hipMalloc((void**)&GPU_Gradientdata, DoubleTotalGPUSize);
    hipMalloc((void**)&GPU_Thetadata, DoubleTotalGPUSize);
	
    // Copy this frame to the GPU:
    hipEventRecord(time1, 0);
    int offsetx, offsety;
	for (int i = 0; i < levels+1; i++) {
		if (i<levels) {
			if (i<levels-1) {
				//printf("\nCurrently on level: %d, Pinned memory offset: %d\n",i,TotalGPUSize/levels*i);
				checkCuda(hipMemcpyAsync(&GPU_idata[TotalGPUSize/levels*i], &CPU_InputArray[TotalGPUSize/levels*i], UCharTotalSize, hipMemcpyHostToDevice, streams[0]), __LINE__); //checkcuda
			}
			else if (i==levels-1) {
				checkCuda(hipMemcpyAsync(&GPU_idata[TotalGPUSize/levels*i], &CPU_InputArray[TotalGPUSize/levels*i], TotalGPUSize/levels, hipMemcpyHostToDevice, streams[0]), __LINE__); //checkcuda
			}
			hipEventRecord(time2,0);
			// Launch kernel:

			offsetx = threadsPerBlock.x*sharedBlocks.x*i;
			for(int j = 0; j<stream; j++) {

				offsety = j*sharedBlocks.y*threadsPerBlock.y;
				GaussianFilter<<<sharedBlocks, threadsPerBlock, 0 , streams[j+1]>>>(GPU_idata, GPU_Gaussdata, M, N, offsetx, offsety);
				checkCuda(hipGetLastError(), __LINE__); //checkcuda
			}
		}
		if (i>0) {
			offsetx = threadsPerBlock.x*sharedBlocks.x*(i-1);
		for(int j = 0; j<stream; j++) {
			offsety = j*sharedBlocks.y*threadsPerBlock.y;
			Sobel<<<sharedBlocks, threadsPerBlock, 0 , streams[j+1]>>>(GPU_Gaussdata, GPU_Gradientdata, GPU_Thetadata, M, N, offsetx, offsety);
			checkCuda(hipGetLastError(), __LINE__); //checkcuda
		}
		for(int j = 0; j<stream; j++) {
			offsety = j*sharedBlocks.y*threadsPerBlock.y;
			Threshold<<<sharedBlocks, threadsPerBlock, 0 , streams[j+1]>>>(GPU_Gradientdata, GPU_Thetadata, GPU_odata, M, N, offsetx, offsety, ThreshLo*8, ThreshLo*8 + 7);
			checkCuda(hipGetLastError(), __LINE__); //checkcuda
		}

		hipEventRecord(time3, 0);
		// Copy result back to CPU:
		//checkCuda( hipMemcpyAsync(CPU_OutputArray, GPU_odata, TotalGPUSize,
		//			   hipMemcpyDeviceToHost, streams[0]) );
		checkCuda(hipMemcpyAsync(&CPU_OutputArray[TotalGPUSize/levels*(i-1)], &GPU_odata[TotalGPUSize/levels*(i-1)], TotalGPUSize/levels, hipMemcpyDeviceToHost, streams[0]), __LINE__); //checkcuda
		hipEventRecord(time4, 0);
		}
	} //end of level loop


  // hipDeviceSynchronize waits for all preceding tasks to finish, and returns
  // an error if any of them failed:
  checkCuda(hipDeviceSynchronize(), __LINE__); //checkcuda

  hipEventSynchronize(time1);
  	hipEventSynchronize(time2);
  	hipEventSynchronize(time3);
  	hipEventSynchronize(time4);

  	float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime;

  	hipEventElapsedTime(&totalTime, time1, time4);
  	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
  	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
  	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

  	Runtimes[0] = totalTime;
  	Runtimes[1] = tfrCPUtoGPU;
  	Runtimes[2] = kernelExecutionTime;
  	Runtimes[3] = tfrGPUtoCPU;

  // Clean up memory:
  for (int i = 0; i < stream+1; ++i) {
  			checkCuda(hipStreamDestroy(streams[i]), __LINE__); //checkcuda
  		  }

    hipFree(GPU_odata);
    hipFree(GPU_idata);
    hipFree(GPU_Gaussdata);
    hipFree(GPU_Gradientdata);
    hipFree(GPU_Thetadata);
  //free(GPU_odata);
  //free(GPU_idata);
    hipEventDestroy(time1);
    	hipEventDestroy(time2);
    	hipEventDestroy(time3);
    	hipEventDestroy(time4);


	ThreshLo++;
	// Done.
	return hipSuccess;
}


	
	
	/**
	
	everything under this needs to be eventually removed
	
	
	uchar **CPU_InputArray = (uchar**) malloc(M * N * sizeof(uchar*));
	uchar **CPU_OutputArray = (uchar**) malloc(M * N * sizeof(uchar*)); //where the GPU should copy the output back to
	if ((CPU_InputArray == NULL) || (CPU_OutputArray == NULL)) {
			fprintf(stderr, "OOPS. Can't create I/O array(s) using malloc() ...\n");
			exit(EXIT_FAILURE);
	}	

		int streamSize = M * N / stream;
		int streamBytes = streamSize * sizeof(uchar);
		int bytes = M * N * sizeof (uchar);

		
		
		//run it


		for (i = 0; i < stream; i++){
			hipStreamCreate(&streams[i]);

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0); // use the first GPU
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	hipEventRecord(time1, streams[i]);

	
	// Allocate GPU buffer for inputs and outputs:
	ucharGPUSize = M * N * sizeof(uchar)/stream;
	doubleGPUSize = M * N * sizeof(double)/stream;
	
	//checkCuda( hipMalloc((void**)&GPU_idata[i], ucharGPUSize) );
	//checkCuda( hipMalloc((void**)&GPU_odata[i], ucharGPUSize) );
	hipMemcpyAsync((void*) GPU_idata[i], &CPU_InputArray[i], ucharGPUSize,
			       hipMemcpyHostToDevice, streams[i]);
	hipMemcpyAsync((void*) GPU_odata[i], zero.data, ucharGPUSize,
			       hipMemcpyHostToDevice, streams[i]);

	hipMalloc((void**)&GPU_Gaussdata[i], doubleGPUSize);
	hipMalloc((void**)&GPU_Gradientdata[i], doubleGPUSize);
	hipMalloc((void**)&GPU_Thetadata[i], doubleGPUSize);

	// Copy input vectors from host memory to GPU buffers.

	hipEventRecord(time2, streams[i]);

	// Launch a kernel on the GPU with one thread for each pixel.
	threadsPerBlock = dim3(BOX_SIZE, BOX_SIZE);
	numBlocks = dim3(M / threadsPerBlock.x/stream, N / threadsPerBlock.y/stream);
	/**GaussianFilter<<<numBlocks, threadsPerBlock>>>(GPU_idata[i], GPU_Gaussdata[i], M, N);

	// Check for errors immediately after kernel launch.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "error code %d (%s) launching gaussian kernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}**/

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	/**cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching addKernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}**/

	// THEN SOBEL THEN THRESHOLD
	/**Sobel<<<numBlocks, threadsPerBlock, 0, streams[i]>>>(GPU_Gaussdata[i], GPU_Gradientdata[i], GPU_Thetadata[i], M, N);

	// Check for errors immediately after kernel launch.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "error code %d (%s) launching sobel kernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	/**cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching addKernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}

	Threshold<<<numBlocks, threadsPerBlock,(size_t) 0,streams[i]>>>(GPU_Gradientdata[i], GPU_Thetadata[i], GPU_odata[i], M, N, ThreshLo*8, ThreshLo*8 + 7);

	// Check for errors immediately after kernel launch.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "error code %d (%s) launching threshold kernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
/**	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching addKernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}
	
	hipEventRecord(time3, streams[i]);
	// Copy output (results) from GPU buffer to host (CPU) memory. 
	hipMemcpyAsync(CPU_OutputArray[i], GPU_odata[i], ucharGPUSize,
			       hipMemcpyDeviceToHost, streams[i]);

	hipEventRecord(time4, streams[i]);
	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

			} //goes up to stream for loop
			
				Error:

	hipFree(GPU_odata);
	hipFree(GPU_idata);
	hipFree(GPU_zerodata);
	hipFree(GPU_Gaussdata);
	hipFree(GPU_Gradientdata);
	hipFree(GPU_Thetadata);

	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);

	printf("!!!!!!!!!!!!!!!!!got here!!!!!!!!!!!!!!!!!after destroy");	
		float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime;
	printf("!!!!!!!!!!!!!!!!!got here!!!!!!!!!!!!!!!!!after time synch");
	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	GPURuntimes[0] = totalTime;
	GPURuntimes[1] = tfrCPUtoGPU;
	GPURuntimes[2] = kernelExecutionTime;
	GPURuntimes[3] = tfrGPUtoCPU;
printf("!!!!!!!!!!!!!!!!!got here!!!!!!!!!!!!!!!!!into gpu time arrays");




	
		ThreshLo++;


		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess){
			fprintf(stderr, "hipDeviceReset failed!\n");
			for (i=0; i<stream; i++) {
				hipHostFree(CPU_InputArray[i]);
				hipHostFree(CPU_OutputArray[i]);
			}
			free(CPU_InputArray);
			free(CPU_OutputArray);
			exit(EXIT_FAILURE);
		}	
		
	//save image to disk
	Mat result = Mat(M,N,CV_8UC1, CPU_OutputArray);
	imwrite(filename,result);

		

		if (!imwrite(filename, result)){
			fprintf(stderr, "couldn't write output to disk!\n");
			for (i=0; i<stream; i++) {
				hipHostFree(CPU_InputArray[i]);
				hipHostFree(CPU_OutputArray[i]);
			}
			free(CPU_InputArray);
			free(CPU_OutputArray);
			exit(EXIT_FAILURE);

		} 
				
		printf("Saved image '%s', size = %dx%d (dims = %d).\n",
			   //filename.c_str(), result.cols, result.rows, result.dims);
			   filename, result.cols, result.rows, result.dims);
		for (i=0; i<stream; i++) {
			hipHostFree(CPU_InputArray[i]);
			hipHostFree(CPU_OutputArray[i]);
		}
		free(CPU_InputArray);
		free(CPU_OutputArray);
	}
	
	exit(EXIT_SUCCESS);
		
}

hipError_t launch_helper(int *CPU_InputArray, int *CPU_OutputArray, float* Runtimes){

	return cudaStatus;
}**/