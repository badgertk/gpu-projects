#include <stdio.h>
#include <stdint.h>
#include <ctype.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

//CUDA STUFF:
#include "hip/hip_runtime.h"
#include ""

//OpenCV stuff
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
using namespace cv;

hipError_t launch_helper(Mat image, int *CPU_OutputArray, float* Runtimes);

#define BOX_SIZE		1 		//ThreadsPerBlock == BOX_SIZE * BOX_SIZE
#define PI				3.1415926
#define EDGE			255
#define NOEDGE			0

int M; //number of rows in the image
int N; //number of columns in the image
int NumIter = 16;
int ThreshLo = 0;
Mat zero;

//ip.Vpixels <--> M
//ip.Hpixels <--> N

//kernels

int main(int argc, char *argv[]){
	float GPURuntimes[4]; //run times of the GPU code
	float ExecTotalTime, TfrCPUGPU, GPUTotalTime, TfrGPUCPU;
	hipError_t cudaStatus;
	char filename[100]; //output file name
	int i;
	int *CPU_OutputArray = (int*) 0; //where the GPU should copy the output back to
	
	if (argc != 3){
		printf("Improper Usage!\n");
		printf("Usage: %s <input image> <output image>\n", argv[0]);
		exit(EXIT_FAILURE);
	}
	ExecTotalTime = 0;
	TfrCPUGPU = 0;
	GPUTotalTime = 0;
	TfrGPUCPU = 0;
	for (i = 0; i < NumIter; i++){

		//Load image:
		Mat image;
		image = imread(argv[1], CV_LOAD_IMAGE_GRAYSCALE);
		if (! image.data){
			fprintf(stderr, "Could not open or find the image.\n");
			exit(EXIT_FAILURE);
		}
		printf("Loaded image '%s', size = %dx%d (dims = %d).\n", argv[1], image.cols, image.rows, image.dims);
		
		//set up global variables for image size
		M = image.rows;
		N = image.cols;
		//Create CPU memory to store the output;
		zero = Mat(M,N,CV_8UC1, Scalar(255)); //start by making every pixel white
		sprintf(filename, "%s%d.png",argv[2],i);
		imwrite(filename, zero);
		
		CPU_OutputArray = (int*) malloc(M*N*sizeof(int));
		if (CPU_OutputArray == NULL){
			fprintf(stderr, "Oops, cannot create CPU_OutputArray using malloc() ...\n");
			exit(EXIT_FAILURE);
		}
		//run it
		cudaStatus = launch_helper(image, CPU_OutputArray, GPURuntimes);
		if (cudaStatus != hipSuccess){
			fprintf(stderr, "launch_helper failed!\n");
			free(CPU_OutputArray);
			exit(EXIT_FAILURE);
		}
		// FIX THIS LAST
		printf("-----------------------------------------------------------------\n");
		printf("Tfr CPU->GPU = %5.2f ms ... \nExecution = %5.2f ms ... \nTfr GPU->CPU = %5.2f ms   \nSum of Iteration = %5.2f ms\n",
				GPURuntimes[1], GPURuntimes[2], GPURuntimes[3], GPURuntimes[0]);
		ExecTotalTime += GPURuntimes[0];
		TfrCPUGPU += GPURuntimes[1];
		GPUTotalTime += GPURuntimes[2];
		TfrGPUCPU += GPURuntimes[3];
		printf("\nTotal Tfr CPU -> GPU Time = %5.2f ms\n", TfrCPUGPU);
		printf("GPU Execution Time = %5.2f ms \n", GPUTotalTime);
		printf("Total Tfr GPU -> CPU Time = %5.2f ms\n", TfrGPUCPU);
		printf("Total Execution Time = %5.2f ms\n", ExecTotalTime);
		printf("-----------------------------------------------------------------\n");
		
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess){
			fprintf(stderr, "hipDeviceReset failed!\n");
			free(CPU_OutputArray);
			exit(EXIT_FAILURE);
		}

		//save image to disk
			Mat result = Mat(M,N,CV_8UC1, CPU_OutputArray);
			imwrite(filename,result);
		

		if (!imwrite(filename, result)){
			fprintf(stderr, "couldn't write output to disk!\n");
			free(CPU_OutputArray);
			exit(EXIT_FAILURE);
		}
		
		printf("Saved image '%s', size = %dx%d (dims = %d).\n",
			   //filename.c_str(), result.cols, result.rows, result.dims);
			   filename, result.cols, result.rows, result.dims);

		free(CPU_OutputArray);
	}
	exit(EXIT_SUCCESS);
		
}

hipError_t launch_helper(Mat image, int *CPU_OutputArray, float* Runtimes){
	
	hipEvent_t time1, time2, time3, time4;
	int ucharGPUSize; // total size of 1 image in bytes
	uchar *GPU_idata;
	uchar *GPU_odata;
	uchar *GPU_zerodata;
	uchar *GPU_Dilationdata;
	uchar *GPU_Erosiondata;
	
	dim3 threadsPerBlock;
	dim3 numBlocks;
	
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0); // use the first GPU
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}
	
	hipEventCreate(&time1);
	hipEventCreate(&time2);
	hipEventCreate(&time3);
	hipEventCreate(&time4);

	hipEventRecord(time1, 0);
	
	// Allocate GPU buffer for inputs and outputs:
	ucharGPUSize = M * N * sizeof(uchar);
	
	cudaStatus = hipMalloc((void**)&GPU_idata, ucharGPUSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&GPU_odata, ucharGPUSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&GPU_zerodata, ucharGPUSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&GPU_Dilationdata, ucharGPUSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&GPU_Erosiondata, ucharGPUSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(GPU_odata, zero.data, ucharGPUSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpyzero failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(GPU_idata, image.data, ucharGPUSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	hipEventRecord(time2, 0);

	// Launch a kernel on the GPU with one thread for each pixel.
	threadsPerBlock = dim3(BOX_SIZE, BOX_SIZE);
	numBlocks = dim3(M / threadsPerBlock.x, N / threadsPerBlock.y);

	//EROSION AND DILATION
	
	Dilation<<<numBlocks, threadsPerBlock>>>(GPU_idata, GPU_Dilationdata, M, N);

	// Check for errors immediately after kernel launch.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "error code %d (%s) launching kernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching addKernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}
	Erosion<<<numBlocks, threadsPerBlock>>>(GPU_idata, GPU_Erosiondata, M, N);

	// Check for errors immediately after kernel launch.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "error code %d (%s) launching kernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching addKernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}
	
	// THEN TAKE THE DIFFERENCE
	
	Difference<<<numBlocks, threadsPerBlock>>>(GPU_Dilationdata, GPU_Erosiondata, GPU_odata, M, N, ThreshLo*8, ThreshLo*8 + 7);

	// Check for errors immediately after kernel launch.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "error code %d (%s) launching kernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching addKernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}
	
	hipEventRecord(time3, 0);
	// Copy output (results) from GPU buffer to host (CPU) memory.
	cudaStatus = hipMemcpy(CPU_OutputArray, GPU_odata, ucharGPUSize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	hipEventRecord(time4, 0);
	
	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime;

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	Runtimes[0] = totalTime;
	Runtimes[1] = tfrCPUtoGPU;
	Runtimes[2] = kernelExecutionTime;
	Runtimes[3] = tfrGPUtoCPU;

	Error:
	hipFree(GPU_odata);
	hipFree(GPU_idata);
	hipFree(GPU_zerodata);
	hipFree(GPU_Dilationdata);
	hipFree(GPU_Erosiondata);
	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);
	ThreshLo++;
	return cudaStatus;
}