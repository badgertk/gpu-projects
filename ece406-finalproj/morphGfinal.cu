#include "hip/hip_runtime.h"
/**
VERSION 3: 	SMALL CHANGES BASED ON VERSION 2 PROVIDED BY FRANCO 12:58 6 DEC
			ATTEMPTED TO FIX LOGICAL ISSUES WITH LEVELS SEE LINE 430(END OF BRACKET AT 443) AND LINE 444(END OF BRACKET AT 487)
			THE FOLLOWING ISSUE WAS FOUND: 
			> CUDA Runtime Error: an illegal memory access was encountered
			
			UPON ONLY RUNNING DILATION AND COMMENTING OUT EROSION AND DIFFERENCE, THE FOLLOWING WAS FOUND:
			> OpenCV Error: Unspecified error (could not find a writer for the specified extension) in imwrite_, file /software/opencv/src/opencv-2.4.11/modules/highgui/src/loadsave.cpp, line 275
			> terminate called after throwing an instance of 'cv::Exception'
			> what():  /software/opencv/src/opencv-2.4.11/modules/highgui/src/loadsave.cpp:275: error: (-2) could not find a writer for the specified extension in function imwrite_



**/

#include <stdio.h>
#include <stdint.h>
#include <ctype.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

//CUDA STUFF:
#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>

//OpenCV stuff
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
using namespace cv;

hipError_t launch_helper(float *CPU_InputArray, float *CPU_OutputArray, float* Runtimes);

inline
hipError_t checkCuda(hipError_t result,int line)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s at line : %d\n", hipGetErrorString(result),line);
    // We should be free()ing CPU+GPU memory here, but we're relying on the OS
    // to do it for us.
    hipDeviceReset();
    assert(result == hipSuccess);
  }
  return result;
}
/*
inline
hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    // We should be free()ing CPU+GPU memory here, but we're relying on the OS
    // to do it for us.
    hipDeviceReset();
    assert(result == hipSuccess);
  }
  return result;
}
*/
#define BOX_SIZE		16 		//ThreadsPerBlock == BOX_SIZE * BOX_SIZE
#define PI				3.1415926
#define EDGE			255
#define NOEDGE			0
#define SHARED_MEM_LIMIT 48*1024

int M; //number of rows in the image
int N; //number of columns in the image
float *CPU_InputArray;
float *CPU_OutputArray; //where the GPU should copy the output back to

char Type;
int Xo; // Width of structuring element
int Yo; // Height of structuring element
uchar * StrElement;
int TotalSize;
int NumIter = 16;
int Thresh = 32;
int nStreams = 4;
int levels = 16;

Mat zero;

//ip.Vpixels <--> M
//ip.Hpixels <--> N

/*__device__ uchar StrElement[3][3] = { 	{	0, 	1, 	0	},
										{	1,	1,	1	},
										{	0,	1,	0	}	};
*/

//kernels

void CreateStrElement() {
	int i, j;
	if (Type=='S') {
		for (i = 0; i<Yo;i++) {
			for (j = 0; j < Xo; j++) {
				StrElement[i*Xo+j] = 1;
			}
		}
	}
	if (Type=='C') {
		int R = sqrt(Xo*Xo/4+Yo*Yo/4);
		int H;
		for (i = -Yo/2; i<Yo/2; i++) {
			H = sqrt(R*R-Yo*Yo/4);
			for (j = -H; j<H; j++) {
				StrElement[(i+Yo/2)*Xo+(j+Xo/2)] = 1;
			}
		}
	}
	if (Type=='X') {
		for (i = 0; i<Yo;i++) {
			if (i==Yo/2) {
				for (j = 0; j < Xo; j++) {
					StrElement[i*Xo+j] = 1;
				}
			}
		}
	}

}

__global__ void Erosion(float *GPU_i, float *Erosiondata, uchar *StrElement, int M, int N, int Xo, int Yo, int offsetx, int offsety){

	extern __shared__ uchar shared_GPU_i[];

	int row = blockIdx.x * blockDim.x + threadIdx.x+offsetx; //row of image
	int col = blockIdx.y * blockDim.y + threadIdx.y+offsety; //column of image
	int idx = row*N + col; //which pixel in full 1D array
	int idy;
	int ids = (threadIdx.x+Yo/2)*blockDim.y+threadIdx.y+Xo/2;
	//uchar output = GPU_i[idx];
	//uchar ElementResult;
	int min;
	int i,j;
	int d = Yo/2;
	int e = Xo/2;
	min = 255;

	if (row >= M || col >= N ) goto End;

	if ((blockIdx.x > 0) && (blockIdx.x < gridDim.x-1)){//if not first row or last row
		if ((blockIdx.y > 0) && (blockIdx.y < gridDim.y-1)){//if not first column or last column (general case)
			if ((threadIdx.x == 0) && (threadIdx.y == 0)){ //top left of block
				for (i=-d; i<=0; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == 0) && (threadIdx.y == (blockDim.y - 1))){ //top right
				for (i=-d; i<=0; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == 0)){ //bottom left
				for (i=0; i<=d; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == (blockDim.y - 1))){ //bottom right
				for (i=0; i<=d; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x > 0) && (threadIdx.x < blockDim.x-1) && (threadIdx.y == 0)) { //left edge
				for (j=-e; j<=0; j++) {
					ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j+e;
					idy = (row)*N+col+j;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.x > 0) && (threadIdx.x < blockDim.x-1) && (threadIdx.y == blockDim.y-1)) { // Right edge
				for (j=0; j<=e; j++) {
					ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j+e;
					idy = (row)*N+col+j;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.y > 0) && (threadIdx.y < blockDim.y-1) && (threadIdx.x == 0)) { //top edge
				for (i=-d; i<=0; i++) {
					ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+e;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.y > 0) && (threadIdx.y < blockDim.y-1) && (threadIdx.x == blockDim.x-1)) { //Bottom edge
				for (i=0; i<=d; i++) {
					ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+e;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			else { //everything else
				ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+e;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
		if (blockIdx.y == 0){//leftmost column - left edge
			if ((threadIdx.x == 0) && (threadIdx.y == (blockDim.y - 1))){ //top right
				for (i=-d; i<=0; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == (blockDim.y - 1))){ //bottom right
				for (i=0; i<=d; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x > 0) && (threadIdx.x < blockDim.x-1) && (threadIdx.y == blockDim.y-1)) { //right edge
				for (j=0; j<=e; j++) {
					ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j;
					idy = (row)*N+col+j;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.y < blockDim.x-1) && (threadIdx.x == 0)) { //top edge
				for (i=-d; i<=0; i++) {
					ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.y < blockDim.x-1) && (threadIdx.x == blockDim.x-1)) { //Bottom edge
				for (i=0; i<=d; i++) {
					ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			else { //everything else
				ids = (threadIdx.x+d)*blockDim.y+threadIdx.y;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
		if (blockIdx.y == (gridDim.y - 1)){//if last column - right edge
			if ((threadIdx.x == 0) && (threadIdx.y == 0)){ //top left of block
				for (i=-d; i<=0; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == 0)){ //bottom left
				for (i=0; i<=d; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x > 0) && (threadIdx.x < blockDim.x-1) && (threadIdx.y == 0)) {//left edge
				for (j=-e; j<=0; j++) {
					ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j+e;
					idy = (row)*N+col+j;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.y > 0) && (threadIdx.y <= blockDim.y-1) && (threadIdx.x == 0)) { //top edge
				for (i=-d; i<=0; i++) {
					ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+e;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.y > 0) && (threadIdx.y <= blockDim.y-1) && (threadIdx.x == blockDim.x-1)) { //Bottom edge
				for (i=0; i<=d; i++) {
					ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+e;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			else { //everything else
				ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+e;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
	}

	if (blockIdx.x == 0){//if first row

		if ((blockIdx.y > 0) && (blockIdx.y < gridDim.y-1)){//if not first column or last column - top edge
			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == 0)){ //bottom left
				for (i=0; i<=d; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == (blockDim.y - 1))){ //bottom right
				for (i=0; i<=d; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x < blockDim.x-1) && (threadIdx.y == 0)) {// Left edge

					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x)*blockDim.y+threadIdx.y+j+e;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.x < blockDim.x-1) && (threadIdx.y == blockDim.y-1) ){ // Right edge
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x)*blockDim.y+threadIdx.y+j+e;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.y > 0) && (threadIdx.y < blockDim.y-1) && (threadIdx.x == blockDim.x-1)) { //bottom edge

				for (i=0; i<=d; i++) {
					ids = (threadIdx.x+i)*blockDim.y+threadIdx.y+e;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			else { //everything else
				ids = (threadIdx.x)*blockDim.y+threadIdx.y+e;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
		if (blockIdx.y == 0){//if first column - top left corner

			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == (blockDim.y - 1))){ //bottom right
				for (i=0; i<=d; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i)*blockDim.y+threadIdx.y+j;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x > 0) && (threadIdx.x < blockDim.y-1) && (threadIdx.y == blockDim.y-1)) { //right edge
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x)*blockDim.y+threadIdx.y+j;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.y < blockDim.y-1) && (threadIdx.x == blockDim.x-1)) { //bottom edge
					for (i=0; i<=d; i++) {
						ids = (threadIdx.x+i)*blockDim.y+threadIdx.y;
						idy = (row+i)*N+col;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			else { //everything else
				ids = (threadIdx.x)*blockDim.y+threadIdx.y;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}

		if (blockIdx.y == (gridDim.y - 1)){//if last column - top right corner

			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == 0)){ //bottom left
				for (i=0; i<=d; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}

			if ((threadIdx.x < blockDim.x-1) && (threadIdx.y == 0)) { // Left edge
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x)*blockDim.y+threadIdx.y+j+e;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}

			}
			if ((threadIdx.y > 0) && (threadIdx.y <= blockDim.y-1) && (threadIdx.x == blockDim.x-1) ){ //Bottom edge

					for (i=0; i<=d; i++) {
						ids = (threadIdx.x+i)*blockDim.y+threadIdx.y+e;
						idy = (row+i)*N+col;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			else { //everything else
				ids = (threadIdx.x)*blockDim.y+threadIdx.y+e;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
	}


	if (blockIdx.x == (gridDim.x - 1)){//if last row
		if ((blockIdx.y > 0) && (blockIdx.y < gridDim.y-1)){//if not first column or last column
			if ((threadIdx.x == 0) && (threadIdx.y == 0)){ //top left of block
				for (i=-d; i<=0; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == 0) && (threadIdx.y == (blockDim.y - 1))){ //top right
				for (i=-d; i<=0; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x > 0) && (threadIdx.x <= blockDim.x-1) && (threadIdx.y == 0)) { // Left edge

					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.x > 0) && (threadIdx.x <= blockDim.x-1) && (threadIdx.y == blockDim.y-1) ){ // Right edge
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.y > 0) && (threadIdx.y < blockDim.y-1) && (threadIdx.x == 0)) { // Top edge
					for (i=-d; i<=0; i++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+e;
						idy = (row+i)*N+col;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			else { //everything else
				ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+e;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}

		if (blockIdx.y == 0){//if first column - bottom left corner
			if ((threadIdx.x == 0) && (threadIdx.y == (blockDim.y - 1))){ //top right
				for (i=-d; i<=0; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}

			if ((threadIdx.x > 0) && (threadIdx.x <= blockDim.x-1) && (threadIdx.y == blockDim.y-1)) { // Right edge
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.y < blockDim.x-1) && (threadIdx.x == 0)) { // Top edge
					for (i=-d; i<=0; i++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y;
						idy = (row+i)*N+col;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			else { //everything else
				ids = (threadIdx.x+d)*blockDim.y+threadIdx.y;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
		if (blockIdx.y == (gridDim.y - 1)){//if last vertical block - bottom right corner
			if ((threadIdx.x == 0) && (threadIdx.y == 0)){ //top left of block
				for (i=-d; i<=0; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}

			if ((threadIdx.x > 0) && (threadIdx.x <= blockDim.x-1) && (threadIdx.y == 0)) { // Left edge
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.y > 0) && (threadIdx.y <= blockDim.y-1) && (threadIdx.x == 0)) { // Top edge
					for (i=-d; i<=0; i++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+e;
						idy = (row+i)*N+col;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			else { //everything else
				ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+e;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
	}

	if ((row < d) || (row > (M - d-1))) goto End;
	int idx2, row2, col2;
	for (i=-d; i<=d; i++){
		for (j=-e; j<=e; j++){
			if (min == 0) continue;
			row2 = row + i;
			col2 = col + j;
			idx2 = row2*N + col2;
			/*if (StrElement[(i+Yo/2)*Xo+j+Xo/2] == 0){
				ElementResult = 255;
				if  (min > ElementResult)
					min = ElementResult;
				//printf("got here 0");
			}*/
			if (StrElement[(i+Yo/2)*Xo+j+Xo/2] == 1){
				//ElementResult = GPU_i[idx2];
				if  (min > shared_GPU_i[idx2])
					min = shared_GPU_i[idx2];
			}
		}
	}

	Erosiondata [idx] = min;
	
	End:;
}

__global__ void Dilation(float *GPU_i, float *Dilationdata, uchar *StrElement, int M, int N, int Xo, int Yo, int offsetx, int offsety){

	extern __shared__ uchar shared_GPU_i[];

	int row = blockIdx.x * blockDim.x + threadIdx.x+offsetx; //row of image
	int col = blockIdx.y * blockDim.y + threadIdx.y+offsety; //column of image
	int idx = row*N + col; //which pixel in full 1D array
	int idy;
	int ids = (threadIdx.x+Yo/2)*blockDim.y+threadIdx.y+Xo/2;
	//uchar output = GPU_i[idx];
	//uchar ElementResult;
	int max;
	int i,j;
	int d = Yo/2;
	int e = Xo/2;
	max = 0;
	if (row >= M || col >= N ) goto End;

	if ((blockIdx.x > 0) && (blockIdx.x < gridDim.x-1)){//if not first row or last row
		if ((blockIdx.y > 0) && (blockIdx.y < gridDim.y-1)){//if not first column or last column (general case)
			if ((threadIdx.x == 0) && (threadIdx.y == 0)){ //top left of block
				for (i=-d; i<=0; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == 0) && (threadIdx.y == (blockDim.y - 1))){ //top right
				for (i=-d; i<=0; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == 0)){ //bottom left
				for (i=0; i<=d; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == (blockDim.y - 1))){ //bottom right
				for (i=0; i<=d; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x > 0) && (threadIdx.x < blockDim.x-1) && (threadIdx.y == 0)) { //left edge
				for (j=-e; j<=0; j++) {
					ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j+e;
					idy = (row)*N+col+j;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.x > 0) && (threadIdx.x < blockDim.x-1) && (threadIdx.y == blockDim.y-1)) { // Right edge
				for (j=0; j<=e; j++) {
					ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j+e;
					idy = (row)*N+col+j;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.y > 0) && (threadIdx.y < blockDim.y-1) && (threadIdx.x == 0)) { //top edge
				for (i=-d; i<=0; i++) {
					ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+e;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.y > 0) && (threadIdx.y < blockDim.y-1) && (threadIdx.x == blockDim.x-1)) { //Bottom edge
				for (i=0; i<=d; i++) {
					ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+e;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			else { //everything else
				ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+e;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
		if (blockIdx.y == 0){//leftmost column - left edge
			if ((threadIdx.x == 0) && (threadIdx.y == (blockDim.y - 1))){ //top right
				for (i=-d; i<=0; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == (blockDim.y - 1))){ //bottom right
				for (i=0; i<=d; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x > 0) && (threadIdx.x < blockDim.x-1) && (threadIdx.y == blockDim.y-1)) { //right edge
				for (j=0; j<=e; j++) {
					ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j;
					idy = (row)*N+col+j;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.y < blockDim.x-1) && (threadIdx.x == 0)) { //top edge
				for (i=-d; i<=0; i++) {
					ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.y < blockDim.x-1) && (threadIdx.x == blockDim.x-1)) { //Bottom edge
				for (i=0; i<=d; i++) {
					ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			else { //everything else
				ids = (threadIdx.x+d)*blockDim.y+threadIdx.y;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
		if (blockIdx.y == (gridDim.y - 1)){//if last column - right edge
			if ((threadIdx.x == 0) && (threadIdx.y == 0)){ //top left of block
				for (i=-d; i<=0; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == 0)){ //bottom left
				for (i=0; i<=d; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x > 0) && (threadIdx.x < blockDim.x-1) && (threadIdx.y == 0)) {//left edge
				for (j=-e; j<=0; j++) {
					ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j+e;
					idy = (row)*N+col+j;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.y > 0) && (threadIdx.y <= blockDim.y-1) && (threadIdx.x == 0)) { //top edge
				for (i=-d; i<=0; i++) {
					ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+e;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			if ((threadIdx.y > 0) && (threadIdx.y <= blockDim.y-1) && (threadIdx.x == blockDim.x-1)) { //Bottom edge
				for (i=0; i<=d; i++) {
					ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+e;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			else { //everything else
				ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+e;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
	}

	if (blockIdx.x == 0){//if first row

		if ((blockIdx.y > 0) && (blockIdx.y < gridDim.y-1)){//if not first column or last column - top edge
			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == 0)){ //bottom left
				for (i=0; i<=d; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == (blockDim.y - 1))){ //bottom right
				for (i=0; i<=d; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x < blockDim.x-1) && (threadIdx.y == 0)) {// Left edge

					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x)*blockDim.y+threadIdx.y+j+e;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.x < blockDim.x-1) && (threadIdx.y == blockDim.y-1) ){ // Right edge
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x)*blockDim.y+threadIdx.y+j+e;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.y > 0) && (threadIdx.y < blockDim.y-1) && (threadIdx.x == blockDim.x-1)) { //bottom edge

				for (i=0; i<=d; i++) {
					ids = (threadIdx.x+i)*blockDim.y+threadIdx.y+e;
					idy = (row+i)*N+col;
					shared_GPU_i[ids]= GPU_i[idy];
				}
			}
			else { //everything else
				ids = (threadIdx.x)*blockDim.y+threadIdx.y+e;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
		if (blockIdx.y == 0){//if first column - top left corner

			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == (blockDim.y - 1))){ //bottom right
				for (i=0; i<=d; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i)*blockDim.y+threadIdx.y+j;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x > 0) && (threadIdx.x < blockDim.y-1) && (threadIdx.y == blockDim.y-1)) { //right edge
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x)*blockDim.y+threadIdx.y+j;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.y < blockDim.y-1) && (threadIdx.x == blockDim.x-1)) { //bottom edge
					for (i=0; i<=d; i++) {
						ids = (threadIdx.x+i)*blockDim.y+threadIdx.y;
						idy = (row+i)*N+col;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			else { //everything else
				ids = (threadIdx.x)*blockDim.y+threadIdx.y;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}

		if (blockIdx.y == (gridDim.y - 1)){//if last column - top right corner

			if ((threadIdx.x == (blockDim.x - 1)) && (threadIdx.y == 0)){ //bottom left
				for (i=0; i<=d; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}

			if ((threadIdx.x < blockDim.x-1) && (threadIdx.y == 0)) { // Left edge
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x)*blockDim.y+threadIdx.y+j+e;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}

			}
			if ((threadIdx.y > 0) && (threadIdx.y <= blockDim.y-1) && (threadIdx.x == blockDim.x-1) ){ //Bottom edge

					for (i=0; i<=d; i++) {
						ids = (threadIdx.x+i)*blockDim.y+threadIdx.y+e;
						idy = (row+i)*N+col;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			else { //everything else
				ids = (threadIdx.x)*blockDim.y+threadIdx.y+e;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
	}


	if (blockIdx.x == (gridDim.x - 1)){//if last row
		if ((blockIdx.y > 0) && (blockIdx.y < gridDim.y-1)){//if not first column or last column
			if ((threadIdx.x == 0) && (threadIdx.y == 0)){ //top left of block
				for (i=-d; i<=0; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x == 0) && (threadIdx.y == (blockDim.y - 1))){ //top right
				for (i=-d; i<=0; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}
			if ((threadIdx.x > 0) && (threadIdx.x <= blockDim.x-1) && (threadIdx.y == 0)) { // Left edge

					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.x > 0) && (threadIdx.x <= blockDim.x-1) && (threadIdx.y == blockDim.y-1) ){ // Right edge
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.y > 0) && (threadIdx.y < blockDim.y-1) && (threadIdx.x == 0)) { // Top edge
					for (i=-d; i<=0; i++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+e;
						idy = (row+i)*N+col;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			else { //everything else
				ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+e;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}

		if (blockIdx.y == 0){//if first column - bottom left corner
			if ((threadIdx.x == 0) && (threadIdx.y == (blockDim.y - 1))){ //top right
				for (i=-d; i<=0; i++) {
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}

			if ((threadIdx.x > 0) && (threadIdx.x <= blockDim.x-1) && (threadIdx.y == blockDim.y-1)) { // Right edge
					for (j=0; j<=e; j++) {
						ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.y < blockDim.x-1) && (threadIdx.x == 0)) { // Top edge
					for (i=-d; i<=0; i++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y;
						idy = (row+i)*N+col;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			else { //everything else
				ids = (threadIdx.x+d)*blockDim.y+threadIdx.y;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
		if (blockIdx.y == (gridDim.y - 1)){//if last vertical block - bottom right corner
			if ((threadIdx.x == 0) && (threadIdx.y == 0)){ //top left of block
				for (i=-d; i<=0; i++) {
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row+i)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
				}
			}

			if ((threadIdx.x > 0) && (threadIdx.x <= blockDim.x-1) && (threadIdx.y == 0)) { // Left edge
					for (j=-e; j<=0; j++) {
						ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+j+e;
						idy = (row)*N+col+j;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			if ((threadIdx.y > 0) && (threadIdx.y <= blockDim.y-1) && (threadIdx.x == 0)) { // Top edge
					for (i=-d; i<=0; i++) {
						ids = (threadIdx.x+i+d)*blockDim.y+threadIdx.y+e;
						idy = (row+i)*N+col;
						shared_GPU_i[ids]= GPU_i[idy];
					}
			}
			else { //everything else
				ids = (threadIdx.x+d)*blockDim.y+threadIdx.y+e;
				idy = (row)*N+col;
				shared_GPU_i[ids]= GPU_i[idy];
			}
		}
	}
	if ((row < d) || (row > (M - d-1))) goto End;
	int idx2, row2, col2;
	for (i=-d; i<=d; i++){
		for (j=-e; j<=e; j++){
			if (max == 255) continue;
			row2 = row + i;
			col2 = col + j;
			idx2 = row2*N + col2;
			/*if (StrElement[(i+Yo/2)*Xo+j+Xo/2] == 0){
				ElementResult = 0;
				if  (max < ElementResult)
					max = ElementResult;
				//printf("got here 0");
			}*/
			if (StrElement[(i+Yo/2)*Xo+j+Xo/2] == 1){
				//ElementResult = GPU_i[idx2];
				if  (max < shared_GPU_i[idx2])
					max = shared_GPU_i[idx2];
			}
		}
	}

	Dilationdata [idx] = max;

	End:;

}


__global__ void Threshold(uchar *GPU_i, uchar *GPU_o, int M, int N, int Thresh, int offsetx, int offsety)
{
    //long tn;            		     // My thread number (ID) is stored here
    //int row,col;
	unsigned char PIXVAL;
	double L,G;

    int rt = blockIdx.x * blockDim.x + threadIdx.x+offsetx;  // row of image
	int ct = blockIdx.y * blockDim.y + threadIdx.y+offsety;  // column of image
	//int k;
	int idx = rt*N+ct;  // which pixel in full 1D array
	if (rt>M-1 || ct>N-1) {
		//GPU_o[idx] = NOEDGE;
		return;
	}

	L=(double)Thresh;		//H=(double)ThreshHi;
	G=GPU_i[idx];
	PIXVAL=NOEDGE;
	if(G<=L){						// no edge
		PIXVAL=NOEDGE;
	}
	else {					// edge
		PIXVAL=EDGE;
	}

	GPU_o[idx]=PIXVAL;

}


__global__ void Difference(float *Dilationdata, float *Erosiondata, float *GPU_o, int M, int N, int offsetx, int offsety){
	int row = blockIdx.x * blockDim.x + threadIdx.x+offsetx; //row of image
	int col = blockIdx.y * blockDim.y + threadIdx.y+offsety; //column of image
	int idx = row*N + col; //which pixel in full 1D array
	int D = Dilationdata[idx];
	int E = Erosiondata[idx];
	GPU_o [idx] = D - E;

}

void show_image(Mat image, string title) {
  //if (1) {
    namedWindow(title, WINDOW_AUTOSIZE);  // create window
    imshow(title, image);                 // show image
    waitKey(0);
  //}
}
	
int main(int argc, char *argv[]){
	float GPURuntimes[4]; //run times of the GPU code
	float ExecTotalTime, TfrCPUGPU, GPUTotalTime, TfrGPUCPU;
	hipError_t cudaStatus;
	//; //output file name
	int i = 1;

	
	if (argc != 6){
		printf("Improper Usage!\n");
		printf("Usage: %s <input image> <output image> <S,X,C> <Width of StrEl> <Height of StrEl>\n", argv[0]);
		printf("Where: S is square-shaped StrEl, X is cross-shaped StrEl, and C is circular StrEl.\n");
		exit(EXIT_FAILURE);
	}
	Type = argv[3][0];
	Xo = atoi(argv[4]);
	Yo = atoi(argv[5]);
	if (Xo*Yo*sizeof(float)>=SHARED_MEM_LIMIT) {
		printf("StrEl element dimensions exceed 48*1024 for float data type. Please make the element smaller.");
		exit(EXIT_FAILURE);
	}

	if (Type == 'C' && Xo != Yo) {
		printf("Error: For circles, StrEl width must equal StrEl height.\n");
		exit(EXIT_FAILURE);
	}

	ExecTotalTime = 0;
	TfrCPUGPU = 0;
	GPUTotalTime = 0;
	TfrGPUCPU = 0;
	//for (i = 0; i < NumIter; i++){

	//Load image:
	Mat image;
	image = imread(argv[1], CV_LOAD_IMAGE_GRAYSCALE);
	if (! image.data){
		fprintf(stderr, "Could not open or find the image.\n");
		exit(EXIT_FAILURE);
	}
	printf("Loaded image '%s', size = %dx%d (dims = %d).\n", argv[1], image.cols, image.rows, image.dims);
	image.convertTo(image, CV_32FC1);
	//set up global variables for image size
	M = image.rows;
	N = image.cols;
	TotalSize = M * N * sizeof(float);
	//Create CPU memory to store the output;
	//zero = Mat(M,N,CV_8UC1, Scalar(255)); //start by making every pixel white
	//sprintf(filename, "%s%d.png",argv[2],i);
	//imwrite(filename, zero);
	checkCuda(hipHostMalloc((void**)&StrElement, Xo*Yo*sizeof(uchar)),__LINE__);
	checkCuda(hipHostMalloc((void**)&CPU_InputArray, TotalSize),__LINE__);
	memcpy(CPU_InputArray, image.data, TotalSize);  // always the same image
	//  Allocate the output while we're at it:
	checkCuda(hipHostMalloc((void**)&CPU_OutputArray, TotalSize),__LINE__);

	CreateStrElement();

	//run it
	checkCuda(launch_helper(CPU_InputArray, CPU_OutputArray, GPURuntimes),__LINE__);
	/*if (cudaStatus != hipSuccess){
		fprintf(stderr, "launch_helper failed!\n");
		free(CPU_OutputArray);
		exit(EXIT_FAILURE);
	}*/
	// FIX THIS LAST
	printf("-----------------------------------------------------------------\n");
	printf("Tfr CPU->GPU = %5.2f ms ... \nExecution = %5.2f ms ... \nTfr GPU->CPU = %5.2f ms   \nSum of Iteration = %5.2f ms\n",
			GPURuntimes[1], GPURuntimes[2], GPURuntimes[3], GPURuntimes[0]);
	/*ExecTotalTime += GPURuntimes[0];
	TfrCPUGPU += GPURuntimes[1];
	GPUTotalTime += GPURuntimes[2];
	TfrGPUCPU += GPURuntimes[3];
	printf("\nTotal Tfr CPU -> GPU Time = %5.2f ms\n", TfrCPUGPU);
	printf("GPU Execution Time = %5.2f ms \n", GPUTotalTime);
	printf("Total Tfr GPU -> CPU Time = %5.2f ms\n", TfrGPUCPU);
	printf("Total Execution Time = %5.2f ms\n", ExecTotalTime);*/
	printf("-----------------------------------------------------------------\n");

	/*cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipDeviceReset failed!\n");
		hipHostFree(StrElement);
		hipHostFree(CPU_InputArray);
		hipHostFree(CPU_OutputArray);
		exit(EXIT_FAILURE);
	}*/

	Mat result = Mat(M, N, CV_32FC1, CPU_OutputArray);

	//save image to disk
	string output_filename = argv[2];
	if (!imwrite(output_filename, image)) {
		fprintf(stderr, "couldn't write output to disk!\n");
		//free();
		exit(EXIT_FAILURE);
	}

	//printf("i : %d\n",i);
	char n0, n1;
	if (i>9) {
		n0 = '1';
		n1 = (i-10)+'0';
		output_filename.insert(output_filename.end()-4,n0);
		output_filename.insert(output_filename.end()-4,n1);
	}

	else {
		n0 = i+'0';
		output_filename.insert(output_filename.end()-4,n0);
	}

	//show_image(result, output_filename);

	//printf("output: %s\n", output_filename.c_str());
	//output_filename[strl-5] = n;
	if (!imwrite(output_filename, result)) {
		fprintf(stderr, "couldn't write output to disk!\n");
		hipHostFree(StrElement);
		hipHostFree(CPU_InputArray);
		hipHostFree(CPU_OutputArray);
		exit(EXIT_FAILURE);
	}
	printf("Saved image '%s', size = %dx%d (dims = %d).\n",
		   output_filename.c_str(), result.rows, result.cols, result.dims);
	hipHostFree(StrElement);
	hipHostFree(CPU_InputArray);
	hipHostFree(CPU_OutputArray);
	checkCuda( hipDeviceReset(), __LINE__ );
	//}
	exit(EXIT_SUCCESS);
		
}

hipError_t launch_helper(float *CPU_InputArray, float *CPU_OutputArray, float* Runtimes){
	
	hipEvent_t time1, time2, time3, time4;
	int ucharGPUSize; // total size of 1 image in bytes
	int sharedMemSize;
	//int offsetx, offsety;
	int TotalSize_2 = (M / levels + Xo/2)*N*sizeof(float);
	float *GPU_idata;
	float *GPU_odata;
	//uchar *GPU_zerodata;
	float *GPU_Dilationdata;
	float *GPU_Erosiondata;
	
	uchar *GPU_StrElement;

	dim3 threadsPerBlock;
	dim3 numBlocks;
	dim3 sharedBlocks;
	dim3 streamSize;
	
	
	hipError_t cudaStatus;
	checkCuda(hipSetDevice(0), __LINE__); // use the first GPU

	hipStream_t stream[nStreams];
	//checkCuda( hipEventCreate(&startEvent) );
	//checkCuda( hipEventCreate(&stopEvent) );
	for (int i = 0; i < nStreams; ++i) {
		checkCuda(hipStreamCreate(&stream[i]),__LINE__);
	}
	

	hipblasHandle_t handle;
	hipblasStatus_t status;
	float alpha = 1;
	float beta = -1;
	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! CUBLAS initialization error\n");
		goto Error;
	}

	threadsPerBlock = dim3(BOX_SIZE, BOX_SIZE);
	numBlocks = dim3(ceil((float)M / threadsPerBlock.x),ceil((float)N / threadsPerBlock.y));
	sharedBlocks = dim3(ceil((float)numBlocks.x / levels), ceil((float)numBlocks.y / nStreams));
	sharedMemSize = (threadsPerBlock.x+Yo)*(threadsPerBlock.y+Xo)*sizeof(float);

	hipEventCreate(&time1);
	hipEventCreate(&time2);
	hipEventCreate(&time3);
	hipEventCreate(&time4);

	hipEventRecord(time1, 0);
	
	// Allocate GPU buffer for inputs and outputs:
	ucharGPUSize = M * N * sizeof(float);
	
	checkCuda( hipMalloc((void**)&GPU_idata, ucharGPUSize), __LINE__);

	checkCuda(hipMalloc((void**)&GPU_odata, ucharGPUSize), __LINE__);
	
	checkCuda(hipMalloc((void**)&GPU_Dilationdata, ucharGPUSize), __LINE__);

	checkCuda( hipMalloc((void**)&GPU_Erosiondata, ucharGPUSize), __LINE__);


	checkCuda( hipMalloc((void**)&GPU_StrElement, Xo*Yo*sizeof(uchar)),__LINE__);

	checkCuda(hipMemset(GPU_Dilationdata, 0, ucharGPUSize),__LINE__);

	checkCuda(hipMemset(GPU_Erosiondata, 0, ucharGPUSize),__LINE__);

	checkCuda(hipMemset(GPU_odata, 0, ucharGPUSize),__LINE__);

	checkCuda(hipMemcpy(GPU_StrElement, StrElement, Xo*Yo*sizeof(uchar), hipMemcpyHostToDevice), __LINE__);
	int offsetx, offsety;
	
	for (int i = 0; i < levels; i++) {
		//if (i < levels + 1) {
			if (i < levels - 1) {
				checkCuda(hipMemcpyAsync(&GPU_idata[M*N / levels*i], &CPU_InputArray[M*N/levels* i], TotalSize_2, hipMemcpyHostToDevice, stream[0]), __LINE__);
			}
			if (i == levels-1) {
				checkCuda(hipMemcpyAsync(&GPU_idata[M*N / levels*i], &CPU_InputArray[M*N/levels*i], ucharGPUSize/levels, hipMemcpyHostToDevice, stream[0]), __LINE__);
			}
			hipEventRecord(time2, 0);
			//printf("Copying levels: %d to %d\n",ucharGPUSize/levels*i, ucharGPUSize/levels*(i+1));
			// Launch a kernel on the GPU with one thread for each pixel.


			//if (i > 0){

			//EROSION AND DILATION
			offsetx = threadsPerBlock.x*sharedBlocks.x*(i);

			for (int j = 0; j < nStreams; j++) {
				offsety = j*sharedBlocks.y*threadsPerBlock.y;
				Dilation << <sharedBlocks, threadsPerBlock, sharedMemSize, stream[j] >> >(GPU_idata, GPU_Dilationdata, GPU_StrElement, M, N, Xo, Yo, offsetx, offsety);

				// Check for errors immediately after kernel launch.
				checkCuda(hipGetLastError(), __LINE__);
			}
				// hipDeviceSynchronize waits for the kernel to finish, and returns
				// any errors encountered during the launch.
			for (int j = 0; j < nStreams; j++){
				offsety = j*sharedBlocks.y*threadsPerBlock.y;
				Erosion << <sharedBlocks, threadsPerBlock, sharedMemSize, stream[j] >> >(GPU_idata, GPU_Erosiondata, GPU_StrElement, M, N, Xo, Yo, offsetx, offsety);

				// Check for errors immediately after kernel launch.
				checkCuda(hipGetLastError(), __LINE__);

				// hipDeviceSynchronize waits for the kernel to finish, and returns
				// any errors encountered during the launch.

			}
		}
		/*if (i > 0) {
			// THEN TAKE THE DIFFERENCE
			offsetx = threadsPerBlock.x*sharedBlocks.x*(i);
			for (int j = 0; j < nStreams; j++){
				offsety = j*sharedBlocks.y*threadsPerBlock.y;
				Difference << <sharedBlocks, threadsPerBlock, 0, stream[j] >> >(GPU_Dilationdata, GPU_Erosiondata, GPU_odata, M, N, offsetx, offsety);

				// Check for errors immediately after kernel launch.
				checkCuda(hipGetLastError(), __LINE__);
			}

			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.

			//checkCuda(hipDeviceSynchronize(),__LINE__);

			hipEventRecord(time3, 0);
			// Copy output (results) from GPU buffer to host (CPU) memory.
			checkCuda(hipMemcpyAsync(&CPU_OutputArray[M*N / levels*(i)],
					&GPU_odata[M*N / levels*(i)], ucharGPUSize / levels,
					hipMemcpyDeviceToHost, stream[0]), __LINE__);
			hipEventRecord(time4, 0);
		//}*/
	status = hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,  // don't transpose either one
	    		       M, N,
	    		       &alpha, GPU_Dilationdata, M,
	    		       &beta,  GPU_Erosiondata, M,
	    		               GPU_odata, M);
		if (status != HIPBLAS_STATUS_SUCCESS)
		{
		fprintf(stderr, "!!!! cuBLAS kernel execution error.\n");
		exit(EXIT_FAILURE);
		}
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		/*cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching addKernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
			goto Error;
		}*/

		/*Threshold <<<numBlocks, threadsPerBlock >>>(GPU_idata, GPU_odata, M, N, Thresh);
		checkCuda( hipGetLastError(), __LINE__ );
	*/
		hipEventRecord(time3, 0);
		// Copy output (results) from GPU buffer to host (CPU) memory.
		status = hipblasGetMatrix(M, N, sizeof(float), GPU_odata, M, CPU_OutputArray, M);
		if (status != HIPBLAS_STATUS_SUCCESS)
		{
		  fprintf(stderr, "!!!! device access error (copying DtoH)\n");
		  exit(EXIT_FAILURE);
		}

	checkCuda(hipDeviceSynchronize(),__LINE__);
	
	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime;

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	Runtimes[0] = totalTime;
	Runtimes[1] = tfrCPUtoGPU;
	Runtimes[2] = kernelExecutionTime;
	Runtimes[3] = tfrGPUtoCPU;
	Error:
	for (int i = 0; i < nStreams; ++i) {
		checkCuda(hipStreamDestroy(stream[i]),__LINE__);
	}
	hipFree(GPU_odata);
	hipFree(GPU_idata);
	hipFree(GPU_Dilationdata);
	hipFree(GPU_Erosiondata);

	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);
	//ThreshLo++;
	return cudaStatus;
}
